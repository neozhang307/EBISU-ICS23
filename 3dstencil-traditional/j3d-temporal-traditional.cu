#include "hip/hip_runtime.h"
#include "./config.cuh"

#include "./common/cuda_computation.cuh"
#include "./common/cuda_common.cuh"
#include "./common/types.hpp"
#include <math.h>

#include <hip/hip_cooperative_groups.h>


#include <cooperative_groups/memcpy_async.h>
#include <cuda_pipeline.h>


namespace cg = cooperative_groups;
// #define isBOX halo
// #define TSTEP (1)

template<class REAL, int halo,  int BLOCKDIM,int LOCAL_TILE_X,int LOCAL_ITEM_PER_THREAD,  int TSTEP>
__global__ void 
kernel3d_temporal_traditional(REAL * __restrict__ input, 
                                REAL *  output, 
                                int width_z, int width_y, int width_x//,
                                // REAL* l2_cache_i, REAL* l2_cache_o
                                )
{
  #define LOCAL_TILE_Y (LOCAL_ITEM_PER_THREAD*BLOCKDIM/LOCAL_TILE_X)
  #define gdim_y (BLOCKDIM/LOCAL_TILE_X)
  #define sizeofsm ((halo*1+1)*(TSTEP)+1)
  #define sizeofreg ((halo+1)*(TSTEP))


  const int tile_x_with_halo=LOCAL_TILE_X+2*halo;
  const int tile_y_with_halo=LOCAL_TILE_Y+2*halo;
  stencilParaT;
  extern __shared__ char sm[];
  REAL* sm_rbuffer = (REAL*)sm+1;

  register REAL r_smbuffer[sizeofreg][REG_Y_SIZE_MOD];

  REAL* smbuffer_buffer_ptr[sizeofsm];
  smbuffer_buffer_ptr[0]=sm_rbuffer;
  #pragma unroll
  for(int hl=1; hl<sizeofsm; hl++)
  {
    smbuffer_buffer_ptr[hl]=smbuffer_buffer_ptr[hl-1]+tile_x_with_halo*tile_y_with_halo;
  }

  const int tid_x = threadIdx.x%LOCAL_TILE_X;
  const int tid_y = threadIdx.x/LOCAL_TILE_X;
  const int index_y = LOCAL_ITEM_PER_THREAD*tid_y;
  const int ps_y = halo;
  const int ps_x = halo;


  const int blocksize_x = LOCAL_TILE_X;
  const int blocksize_y = LOCAL_TILE_Y;
  const int valid_blocksize_x = blocksize_x-TSTEP*halo*2;
  const int p_x_real = blockIdx.x * valid_blocksize_x - TSTEP*halo;
  const int valid_blocksize_y = blocksize_y-TSTEP*halo*2;
  const int p_y_real = blockIdx.y * valid_blocksize_y - TSTEP*halo;

  const int p_x = p_x_real;//blockIdx.x * LOCAL_TILE_X;
  const int p_y = p_y_real;//blockIdx.y * LOCAL_TILE_Y;


  int blocksize_z=((width_z+gridDim.z-1)/gridDim.z);
  const int p_z =  blockIdx.z * (blocksize_z);
  const int p_z_end =  p_z + (blocksize_z); 

  const int p_z_real     = p_z-(halo+1)*(TSTEP-1);//-((halo*LOCAL_STEP+LOCAL_TILE_Y-1)/LOCAL_TILE_Y)*LOCAL_TILE_Y-halo;//(LOCAL_TILE_Y)*(LOCAL_STEP-1);//(p_y - LOCAL_STEP*(halo)*2);
  const int p_z_real_end = p_z_end+(halo+1)*(TSTEP-1); //(p_y_end + LOCAL_STEP*(halo+LOCAL_TILE_Y));

  {
    for(int l_z=0; l_z<halo+1; l_z++)
    {
      for(int lid=threadIdx.x; lid<tile_x_with_halo*tile_y_with_halo; lid+=blockDim.x)
      {
        int l_x=lid%tile_x_with_halo-halo;
        int l_y=lid/tile_x_with_halo-halo;

        // int l_global_z=(MIN(p_z-2*halo*(TSTEP-1)+l_z,width_z-1));
        int l_global_z=(MIN(p_z_real+l_z,width_z-1));
        l_global_z=(MAX(l_global_z,0));

        int l_global_y = (MIN(p_y+l_y,width_y-1));
          l_global_y = (MAX(l_global_y,0));
        int l_global_x = (MIN(p_x+l_x,width_x-1));
          l_global_x = (MAX(l_global_x,0));
        __pipeline_memcpy_async(smbuffer_buffer_ptr[l_z+sizeofsm-1-halo-1]+tile_x_with_halo*(l_y+ps_y)+l_x+ps_x, 
              input+l_global_z*width_x*width_y+l_global_y*width_x+l_global_x , sizeof(REAL));
      } 
    }

    __pipeline_commit();
    __pipeline_wait_prior(0);
    __syncthreads();

    _Pragma("unroll")
    for(int l_y=0; l_y<LOCAL_ITEM_PER_THREAD; l_y++)
    {
      _Pragma("unroll")
      for(int l_z=0; l_z<halo ; l_z++)
      {
        int l_global_z=(MIN(p_z_real+l_z-halo,width_z-1));
        l_global_z=(MAX(l_global_z,0));

        int l_global_y = (MIN(l_y+index_y+p_y,width_y-1));
          l_global_y = (MAX(l_global_y,0));
        int l_global_x = (MIN(p_x+tid_x,width_x-1));
          l_global_x = (MAX(l_global_x,0));

        r_smbuffer[l_z+sizeofreg-(halo+1)][l_y] = input[l_global_z*width_x*width_y+l_global_y*width_x+l_global_x];// smbuffer_buffer_ptr[l_z+sizeofsm-(2*halo+2)][(l_y+ps_y+index_y)*tile_x_with_halo+ps_x+tid_x];//input[(global_y) * width_x + global_x];
      }
    }

    
    for(int global_z=p_z_real; global_z<p_z_real_end; global_z+=1)
    {
      //preload next step
      if(global_z<p_z_end+(halo+1)*(TSTEP-1))
      {

        int l_global_z=(MIN(global_z+halo+1,width_z-1));
        l_global_z=(MAX(l_global_z,0));
        for(int lid=threadIdx.x; lid<tile_x_with_halo*tile_y_with_halo; lid+=blockDim.x)
        {
          int l_x=lid%tile_x_with_halo-halo;
          int l_y=lid/tile_x_with_halo-halo;

          int l_global_y = (MIN(p_y+l_y,width_y-1)); 
            l_global_y = (MAX(l_global_y,0));
          int l_global_x = (MIN(p_x+l_x,width_x-1));
            l_global_x = (MAX(l_global_x,0));

          __pipeline_memcpy_async(smbuffer_buffer_ptr[sizeofsm-1]+tile_x_with_halo*(l_y+ps_y)+l_x+ps_x, 
                input+l_global_z*width_x*width_y+l_global_y*width_x+l_global_x , sizeof(REAL));
        }
      }
      __pipeline_commit();
      // might not be ble to unroll
      // __syncthreads();
      REAL sum[TSTEP][LOCAL_ITEM_PER_THREAD];

        //sm2reg
      //need to unroll
      _Pragma("unroll")
      for(int step=1; step<TSTEP; step++)
      {
        // REAL sum[step][LOCAL_ITEM_PER_THREAD];
        _Pragma("unroll")
        for(int l_y=0; l_y<LOCAL_ITEM_PER_THREAD; l_y++)
        {
          sum[step][l_y]=0;
        }

        _Pragma("unroll")
        for(int l_y=0; l_y<LOCAL_ITEM_PER_THREAD; l_y++)
        {
          r_smbuffer[sizeofreg-step*(halo+1)+halo][l_y] = smbuffer_buffer_ptr[sizeofsm-(step)*(1*halo+1)-1][(ps_y+index_y+l_y)*tile_x_with_halo+ps_x+tid_x];
        }
        //main computation
        computation<REAL,LOCAL_ITEM_PER_THREAD,halo,REG_Y_SIZE_MOD>(sum[step],
                                          smbuffer_buffer_ptr+sizeofsm-(step)*(1*halo+1)-1,
                                          ps_y+index_y, tile_x_with_halo, tid_x+ps_x,
                                          r_smbuffer+sizeofreg-step*(halo+1),
                                          stencilParaInput,isBOX);
      }
        //star version can use multi-buffer to remove the necessarity of two sync
      __syncthreads();
      _Pragma("unroll")
      for(int step=1; step<TSTEP; step++)
      {
        //the following part only onlly need to store boundary
        _Pragma("unroll")
        for(int l_y=0; l_y<LOCAL_ITEM_PER_THREAD; l_y++)
        {
          smbuffer_buffer_ptr[sizeofsm-(step)*(halo+1)-1][(ps_y+index_y+l_y)*tile_x_with_halo+ps_x+tid_x]=sum[step][l_y];
        }
      }
      // __syncthreads();


      _Pragma("unroll")
      for(int l_y=0; l_y<LOCAL_ITEM_PER_THREAD; l_y++)
      {
        sum[0][l_y]=0;
      }
      _Pragma("unroll")
      for(int l_y=0; l_y<LOCAL_ITEM_PER_THREAD; l_y++)
      {
          r_smbuffer[halo][l_y] = smbuffer_buffer_ptr[0][(ps_y+index_y+l_y)*tile_x_with_halo+ps_x+tid_x];
      }
      //main computation
      computation<REAL,LOCAL_ITEM_PER_THREAD,halo,REG_Y_SIZE_MOD>(sum[0],
                                        smbuffer_buffer_ptr,
                                        ps_y+index_y, tile_x_with_halo, tid_x+ps_x,
                                        r_smbuffer,
                                        stencilParaInput,isBOX);
      
      //star version can use multi-buffer to remove the necessarity of two sync
      // // reg 2 ptr
      int global_z2=global_z-(halo+1)*(TSTEP-1);
      // int global_z2=global_z-(2*halo+1)*(TSTEP-1);
      if(tid_x>=halo*TSTEP&&tid_x<blocksize_x-halo*TSTEP)
      {
        _Pragma("unroll")
        for(int l_y=0; l_y<LOCAL_ITEM_PER_THREAD; l_y++)
        {
          if(p_x+tid_x>=width_x)break;
          if(p_y+index_y+l_y>=width_y)break;

          if(index_y+l_y>=blocksize_y-halo*TSTEP)break;
          if(index_y+l_y<halo*TSTEP)continue;

          if(global_z2<p_z)break;
          if(global_z2>=p_z_end)break;
          if(global_z2>=width_z)break;
          output   [global_z2*width_x*width_y+(p_y+index_y+l_y)*width_x+p_x+tid_x]=sum[0][l_y];
        }
      }
      


      // __syncthreads();
      // gg.sync();
      
       REAL* tmp = smbuffer_buffer_ptr[0];
      // smswap 
      _Pragma("unroll")
      for(int hl=1; hl<sizeofsm; hl++)
      {
        smbuffer_buffer_ptr[hl-1]=smbuffer_buffer_ptr[hl];
      }
      smbuffer_buffer_ptr[sizeofsm-1]=tmp;
  //  __syncthreads();
      _Pragma("unroll")
      for(int l_z=0; l_z<TSTEP; l_z++)
      {
        _Pragma("unroll")
        for(int l_h=0; l_h<HALO; l_h++)
        {
          _Pragma("unroll")
          for(int l_y=0; l_y<REG_Y_SIZE_MOD; l_y++)
          {
          
            r_smbuffer[l_z*(halo+1)+l_h][l_y]=r_smbuffer[l_z*(halo+1)+1+l_h][l_y];
          }
        }
      }

      // REAL* tmp_ptr =l2_cache_i;
      // l2_cache_i=l2_cache_o;
      // l2_cache_o=tmp_ptr;

      __pipeline_wait_prior(0);
      __syncthreads();
    }

  }
  #undef LOCAL_TILE_X
  #undef gdim_y
  #undef sizeofsm
  #undef sizeofreg
}

template __global__ void kernel3d_temporal_traditional<double,HALO>(double*__restrict__,double*,int,int,int);
template __global__ void kernel3d_temporal_traditional<float,HALO>(float*__restrict__,float*,int,int,int);
